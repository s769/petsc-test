#include "shared.cuh"

int main(int argc, char **argv) {


    int world_rank, num_ranks;
    int proc_rows, proc_cols;
    bool prflag, pcflag;
    int nt, nm;
    bool nflag;
    PetscFunctionBeginUser;
    PetscCall(PetscInitialize(&argc, &argv, NULL, NULL));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-proc_rows", &proc_rows, NULL));
    // PetscCall(PetscCheck(prflag, PETSC_COMM_WORLD, PETSC_ERR_USER, "Must specify -proc_rows"));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-proc_cols", &proc_cols, NULL));
    // PetscCall(PetscCheck(pcflag,  PETSC_COMM_WORLD, PETSC_ERR_USER, "Must specify -proc_cols"));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-nt", &nt, NULL));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-nm", &nm, NULL));

    // PetscCall(PetscCheck(nflag,  PETSC_COMM_WORLD, PETSC_ERR_USER, "Must specify -n"));


    PetscCall(MPI_Comm_rank(MPI_COMM_WORLD, &world_rank));
    PetscCall(MPI_Comm_size(MPI_COMM_WORLD, &num_ranks));

    int row_rank = world_rank / proc_cols;
    int col_rank = world_rank % proc_cols;
    double *a, *d_a;
    int nm_local = (col_rank < nm % proc_cols) ? nm / proc_cols + 1 : nm / proc_cols;

    int before_me = (col_rank < nm % proc_cols) ? (nm/proc_cols + 1) * col_rank : (nm/proc_cols + 1) * nm % proc_cols + (nm/proc_cols) * (col_rank - nm % proc_cols);

    Vec v;
    if (row_rank == 0)
    {
        a = (double *) malloc(nm_local * nt * sizeof(double));
        for (int i = 0; i < nm_local; i++) {
            for (int j = 0; j < nt; j++) {
                a[i*nt + j] = before_me*nt + i*nt + j;
            }
        }
        hipMalloc(&d_a, nm_local * nt * sizeof(double));
        hipMemcpy(d_a, a, nm_local * nt * sizeof(double), hipMemcpyHostToDevice);
        free(a);

    }

    PetscCall(VecCreate(PETSC_COMM_WORLD, &v));
    PetscLayout layout;
    PetscCall(PetscLayoutCreate(PETSC_COMM_WORLD, &layout));
    PetscCall(PetscLayoutSetBlockSize(layout, 1));
    if (row_rank == 0)
        PetscCall(PetscLayoutSetLocalSize(layout, nm_local * nt));
    else
        PetscCall(PetscLayoutSetLocalSize(layout, 0));
    PetscCall(PetscLayoutSetUp(layout));
    PetscCall(VecSetLayout(v, layout));
    PetscCall(VecSetType(v, VECCUDA));


    if (row_rank == 0)
    {
        PetscCall(VecCUDAReplaceArray(v, d_a));
    }
    else
    {
        PetscCall(VecCUDAReplaceArray(v, NULL));
    }
    PetscCall(VecSetUp(v));


    PetscCall(VecView(v, PETSC_VIEWER_STDOUT_WORLD));

    VecScatter scatter;
    Vec v2;
    IS is;
    
    PetscLayout layout2;
    PetscCall(PetscLayoutCreate(PETSC_COMM_WORLD, &layout2));
    PetscCall(PetscLayoutSetBlockSize(layout2, 1));
    int nt_local = (world_rank < nt % num_ranks) ? nt / num_ranks + 1 : nt / num_ranks;
    int before_me2 = (world_rank < nt % num_ranks) ? (nt/num_ranks + 1) * world_rank : (nt/num_ranks + 1) * (nt % num_ranks) + (nt/num_ranks) * (world_rank - nt % num_ranks);

    printf("world_rank: %d, nt_local: %d, before_me2: %d\n", world_rank, nt_local, before_me2);
    if (col_rank == 0)
        PetscCall(PetscLayoutSetLocalSize(layout2, nt_local * nm));
    else
        PetscCall(PetscLayoutSetLocalSize(layout2, 0));
    
    PetscCall(PetscLayoutSetUp(layout2));
    PetscCall(VecCreate(PETSC_COMM_WORLD, &v2));
    PetscCall(VecSetLayout(v2, layout2));
    PetscCall(VecSetType(v2, VECCUDA));
    PetscCall(VecSetUp(v2));

    int* idx = new int[nt_local * nm];

    for (int i = 0; i < nt_local; i++)
    {
        for (int j = 0; j < nm; j++)
        {
            idx[i*nm + j] = before_me2 + i + j*nt;
            printf("world_rank: %d, idx: %d\n", world_rank, idx[i*nm + j]);
        }
    }



    delete[] idx;

    PetscCall(VecDestroy(&v));
    PetscCall(PetscLayoutDestroy(&layout));





    

    PetscCall(PetscFinalize());
    return 0;

}