#include "shared.cuh"

int main(int argc, char **argv) {


    int world_rank, num_ranks;
    int proc_rows, proc_cols;
    bool prflag, pcflag;
    int nt, nm;
    bool nflag;
    PetscFunctionBeginUser;
    PetscCall(PetscInitialize(&argc, &argv, NULL, NULL));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-proc_rows", &proc_rows, NULL));
    // PetscCall(PetscCheck(prflag, PETSC_COMM_WORLD, PETSC_ERR_USER, "Must specify -proc_rows"));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-proc_cols", &proc_cols, NULL));
    // PetscCall(PetscCheck(pcflag,  PETSC_COMM_WORLD, PETSC_ERR_USER, "Must specify -proc_cols"));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-nt", &nt, NULL));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-nm", &nm, NULL));

    // PetscCall(PetscCheck(nflag,  PETSC_COMM_WORLD, PETSC_ERR_USER, "Must specify -n"));


    PetscCall(MPI_Comm_rank(MPI_COMM_WORLD, &world_rank));
    PetscCall(MPI_Comm_size(MPI_COMM_WORLD, &num_ranks));

    int row_rank = world_rank / proc_cols;
    int col_rank = world_rank % proc_cols;
    double *a, *d_a;
    int nm_local = (col_rank == proc_cols - 1) ? nm / proc_cols + nm % proc_cols : nm / proc_cols;


    Vec v;
    if (row_rank == 0)
    {
        a = (double *) malloc(nm_local * nt * sizeof(double));
        for (int i = 0; i < nm_local; i++) {
            for (int j = 0; j < nt; j++) {
                a[i*nt + j] = world_rank*(nm/proc_cols)*nt + i*nt + j + 1;
                printf("i * nt + j = %d, world_rank*(nm/proc_cols)*nt + i*nt + j + 1: %d\n", i*nt + j, world_rank*(nm/proc_cols)*nt + i*nt + j + 1);
        
            }
        }
        hipMalloc(&d_a, nm_local * nt * sizeof(double));
        hipMemcpy(d_a, a, nm_local * nt * sizeof(double), hipMemcpyHostToDevice);
        free(a);

    }

    PetscCall(VecCreate(PETSC_COMM_WORLD, &v));
    PetscLayout layout;
    PetscCall(PetscLayoutCreate(PETSC_COMM_WORLD, &layout));
    PetscCall(PetscLayoutSetBlockSize(layout, 1));
    if (row_rank == 0)
        PetscCall(PetscLayoutSetLocalSize(layout, nm_local));
    else
        PetscCall(PetscLayoutSetLocalSize(layout, 0));
    PetscCall(PetscLayoutSetUp(layout));
    PetscCall(VecSetLayout(v, layout));
    PetscCall(VecSetType(v, VECCUDA));


    if (row_rank == 0)
    {
        PetscCall(VecCUDAReplaceArray(v, d_a));
    }
    else
    {
        PetscCall(VecCUDAReplaceArray(v, NULL));
    }
    PetscCall(VecSetUp(v));


    PetscCall(VecView(v, PETSC_VIEWER_STDOUT_WORLD));

    // VecScatter scatter;
    // IS is;
    
    // PetscLayout layout2;
    // PetscCall(PetscLayoutCreate(PETSC_COMM_WORLD, &layout2));
    // PetscCall(PetscLayoutSetBlockSize(layout2, 1));
    // nm_local2 = 
    






    PetscCall(VecDestroy(&v));
    PetscCall(PetscLayoutDestroy(&layout));





    

    PetscCall(PetscFinalize());
    return 0;

}