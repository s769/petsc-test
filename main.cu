#include "hip/hip_runtime.h"
#include "shared.cuh"

int main(int argc, char **argv) {


    int world_rank, num_ranks;
    int proc_rows, proc_cols;
    bool prflag, pcflag;
    int n;
    bool nflag;
    PetscFunctionBeginUser;
    PetscCall(PetscInitialize(&argc, &argv, NULL, NULL));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-proc_rows", &proc_rows, &prflag));
    PetscCall(PetscCheck(prflag, "Must specify -proc_rows"));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-proc_cols", &proc_cols, &pcflag));
    PetscCall(PetscCheck(pcflag, "Must specify -proc_cols"));
    PetscCall(PetscOptionsGetInt(NULL, NULL, "-n", &n, &nflag));
    PetscCall(PetscCheck(nflag, "Must specify -n"));


    PetscCall(MPI_Comm_rank(MPI_COMM_WORLD, &world_rank));
    PetscCall(MPI_Comm_size(MPI_COMM_WORLD, &num_ranks));

    int row_rank = world_rank / proc_cols;
    int col_rank = world_rank % proc_cols;
    double *a, *d_a;

    Vec v;
    if (row_rank == 0)
    {
        a = (double *) malloc(n * sizeof(double));
        for (int i = 0; i < n; i++) {
            a[i] = i;
        }
        hipMalloc(&d_a, n * sizeof(double));
        hipMemcpy(d_a, a, n * sizeof(double), hipMemcpyHostToDevice);
        free(a);

    }

    PetscCall(VecCreate(PETSC_COMM_WORLD, &v));
    PetscLayout layout;
    PetscCall(PetscLayoutCreate(PETSC_COMM_WORLD, &layout));
    PetscCall(PetscLayoutSetBlockSize(layout, 1));
    if (row_rank == 0)
        PetscCall(PetscLayoutSetLocalSize(layout, n/proc_cols));
    else
        PetscCall(PetscLayoutSetLocalSize(layout, 0));
    PetscCall(PetscLayoutSetUp(layout));
    PetscCall(VecSetLayout(v, layout));
    PetscCall(VecSetType(v, VECCUDA));


    if (row_rank == 0)
    {
        PetscCall(VecCUDAReplaceArray(v, d_a));
    }
    PetscCall(VecSetUp(v));

    printf("hello from rank %d\n", world_rank);

    PetscCall(VecView(v, PETSC_VIEWER_STDOUT_WORLD));

    Vec w;
    PetscCall(VecDuplicate(v, &w));
    PetscCall(VecZeroEntries(w));

    PetscCall(VecView(w, PETSC_VIEWER_STDOUT_WORLD));

    PetscCall(VecAXPY(w, 1.0, v)); 


    double norm;
    PetscCall(VecNorm(v, NORM_2, &norm)); // replace with norm of w and doesnt hang
    PetscPrintf(PETSC_COMM_WORLD, "Norm: %f\n", norm);

    PetscCall(VecDestroy(&v));
    PetscCall(VecDestroy(&w));
    PetscCall(PetscLayoutDestroy(&layout));


    

    PetscCall(PetscFinalize());
    return 0;

}